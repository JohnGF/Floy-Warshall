#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <time.h>

#define GRAPH_SIZE 2000
#define EDGE_COST(graph, graph_size, a, b) graph[a * graph_size + b]
#define D(a, b) EDGE_COST(output, graph_size, a, b)
#define INF 0x1fffffff

int dis = 0;
int k = 4;

void generate_random_graph(int* output, int graph_size) {
    int i, j;

    srand(0xdadadada);

    for (i = 0; i < graph_size; i++) {
        for (j = 0; j < graph_size; j++) {
            if (i == j) {
                D(i, j) = 0;
            }
            else {
                int r;
                r = rand() % 40;
                if (r > 20) {
                    r = INF;
                }

                D(i, j) = r;
            }
        }
    }
}
__global__
void floyd_warshall_cpu(const int* graph, int graph_size, int* output) {
    int i, j, k,t;

    memcpy(output, graph, sizeof(int) * graph_size * graph_size);

    for (k = 0; k < graph_size; k++) {
        for (i = 0; i < graph_size; i++) {
            for (j = 0; j < graph_size; j++) {

                t = D(i, k) + D(k, j);
                D(i, j) = t * (t < D(i, j)) + D(i, j) * (t >= D(i, j));

            }
        }
    }
}

void floyd_warshall_gpu(const int* graph, int graph_size, int* output) {
    int i, j, k;

    memcpy(output, graph, sizeof(int) * graph_size * graph_size);

    for (k = 0; k < graph_size; k++) {
        for (i = 0; i < graph_size; i++) {
            for (j = 0; j < graph_size; j++) {
                if (D(i, k) + D(k, j) < D(i, j)) {
                    D(i, j) = D(i, k) + D(k, j);
                }
            }
        }
    }

}
int main(int argc, char** argv) {
    int* graph, * output_cpu, * output_gpu;
    int size;

    size = sizeof(int) * GRAPH_SIZE * GRAPH_SIZE;

    graph = (int*)malloc(size);
    assert(graph);

    output_cpu = (int*)malloc(size);
    assert(output_cpu);
    memset(output_cpu, 0, size);

    output_gpu = (int*)malloc(size);
    assert(output_gpu);

    generate_random_graph(graph, GRAPH_SIZE);
    fprintf(stderr, "running on cpu...\n");
    floyd_warshall_cpu(graph, GRAPH_SIZE, output_cpu);
    fprintf(stderr, "running on gpu...\n");
    floyd_warshall_gpu(graph, GRAPH_SIZE, output_gpu);
    bool works = output_cpu == output_gpu;
    //fprintf("%d\n",output_cpu);
    //fprintf("%d\n",output_gpu);
    printf("%d\n", works);

    if (memcmp(output_cpu, output_gpu, size) != 0) {
        fprintf(stderr, "FAIL!\n");
    }
    return 0;
}