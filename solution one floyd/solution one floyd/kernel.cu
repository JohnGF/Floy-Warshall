﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <time.h>

#include <cstdio>
#include <cassert>
#include <stdlib.h>

#include <string.h>


#define GRAPH_SIZE 2000
//macro for adding a edge_cost to cell
#define EDGE_COST(graph, graph_size, a, b) graph[a * graph_size + b]
//macro for getting edgE_cost 
#define D(a, b) EDGE_COST(output, graph_size, a, b)
//macro to have type 
#define INF 0x1fffffff

#define THREADS_PER_BLOCK_SIDE 16
#define BLOCKS_PER_GRAPH_SIDE ((GRAPH_SIZE+THREADS_PER_BLOCK_SIDE-1) / THREADS_PER_BLOCK_SIDE)
#define HANDLE_ERROR(x) x



__global__ void run_on_gpu(const int graph_size, int* output, int k) {
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    if ((i < graph_size) && (j < graph_size))
        if (D(i, k) + D(k, j) < D(i, j)) {
            D(i, j) = D(i, k) + D(k, j);
        }
}

void floyd_warshall_gpu(const int* graph, int graph_size, int* output) {
    int* dev_output;

    HANDLE_ERROR(hipMalloc(&dev_output, sizeof(int) * graph_size * graph_size));

    hipMemcpy(dev_output, graph, sizeof(int) * graph_size * graph_size, hipMemcpyHostToDevice);
    dim3 blocks(BLOCKS_PER_GRAPH_SIDE, BLOCKS_PER_GRAPH_SIDE, 1);
    dim3 threadsPerBlock(THREADS_PER_BLOCK_SIDE, THREADS_PER_BLOCK_SIDE, 1);
    int k;
    for (k = 0; k < graph_size; k++) {
        run_on_gpu <<<blocks, threadsPerBlock >>> (graph_size, dev_output, k);
    }
    hipMemcpy(output, dev_output, sizeof(int) * graph_size * graph_size, hipMemcpyDeviceToHost);

    hipFree(dev_output);
}

void generate_random_graph(int* output, int graph_size) {
    int i, j;

    srand(0xdadadada);

    for (i = 0; i < graph_size; i++) {
        for (j = 0; j < graph_size; j++) {
            if (i == j) {
                D(i, j) = 0;
            }
            else {
                int r;
                r = rand() % 1000;
                if (r > 20) {
                    D(i, j) = INF;
                }
                else
                    D(i, j) = r + 10;
            }
        }
    }
}

void floyd_warshall_cpu(const int* graph, int graph_size, int* output) {
    int i, j, k;

    memcpy(output, graph, sizeof(int) * graph_size * graph_size);

    for (k = 0; k < graph_size; k++) {
        for (i = 0; i < graph_size; i++) {
            for (j = 0; j < graph_size; j++) {
                if (D(i, k) + D(k, j) < D(i, j)) {
                    D(i, j) = D(i, k) + D(k, j);
                }
            }
        }
    }
}

int main(int argc, char** argv) {
    clock_t start, end;
    int* graph, * output_cpu, * output_gpu;
    int size;
    size = sizeof(int) * GRAPH_SIZE * GRAPH_SIZE;
    //returns a pointer to memory adress for the graph
    graph = (int*)malloc(size);
    output_cpu = (int*)malloc(size);
    assert(output_cpu);
    memset(output_cpu, 0, size);
    output_gpu = (int*)malloc(size);
    generate_random_graph(graph, GRAPH_SIZE);
    start = clock();
    floyd_warshall_cpu(graph, GRAPH_SIZE, output_cpu);
    end = clock();
    double duration = ((double)end - start) / CLOCKS_PER_SEC;
    printf("Time taken to execute cpu in seconds : %f\n", duration);
    start = clock();
    floyd_warshall_gpu(graph, GRAPH_SIZE, output_gpu);
    end = clock();
    duration = ((double)end - start) / CLOCKS_PER_SEC;
    printf("Time taken to execute gpu in seconds : %f\n", duration);

    if (memcmp(output_cpu, output_gpu, size) != 0) {
        fprintf(stderr, "FAIL!\n");
        int qq = 0;
        //tries to run mismatch in the matrix
        for (int i = 0; i < GRAPH_SIZE * GRAPH_SIZE; i++)
            if (output_cpu[i] != output_gpu[i]) { qq++; printf("i: %d, cpu: %d, gpu: %d\n", i, output_cpu[i], output_gpu[i]); }
        printf("# mismatches: %d\n", qq);
    }
    else {
        fprintf(stderr, "SUCCESS!\n");
        //  for (int i = 0; i < 100; i++)
        //   printf("i: %d, cpu: %d, gpu: %d\n",i, output_cpu[i], output_gpu[i]);
    }
    //free memory
    free(graph);
    free(output_cpu);
    free(output_gpu);
    return 0;
}